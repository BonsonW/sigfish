#include "hip/hip_runtime.h"

#include "sigfish.h"
#include "cdtw.h"
#include "cudtw.cuh"
#include "assert.h"

#define MALLOC_CHK(ret) { \
    if ((ret) == NULL) { \
        fprintf(stderr,"Could not allocate memory."); \
    } \
}

aln_t *init_aln2(){
    aln_t *aln = (aln_t *)malloc(sizeof(aln_t)*SECONDARY_CAP);
    MALLOC_CHK(aln);
    float score = INFINITY;
    float score2 = INFINITY;
    int32_t pos = -1;
    int32_t rid = -1;
    char d = 0;
    for (int l=0; l<SECONDARY_CAP;l++) {
        aln_t tmp = {rid,pos,pos,score,score2,d,0};
        aln[l] = tmp;
    }

    return aln;
}


void update_aln2(aln_t* aln, float score, int32_t rid, int32_t pos, char d, float *cost, int32_t qlen, int32_t rlen){
    int l=0;
    for(; l<SECONDARY_CAP; l++){
        if (score > aln[l].score){
            break;
        } else {
            continue;
        }
    }

    if(l!=0){
        for(int m=0;m<l-1;m++){
            aln[m] = aln[m+1];
        }
        aln[l-1].score = score;
        aln[l-1].pos_end = pos;
        aln[l-1].rid = rid;
        aln[l-1].d = d;

        aln[l-1].pos_st  = pos - qlen + 1;

    }
}

void dtw_single2(core_t* core,db_t* db, int32_t i) {

    if(db->slow5_rec[i]->len_raw_signal>0 && db->et[i].n>0){ //some checks to see if a good read

        aln_t *aln=init_aln2(); //initialise a alignment struct

        int64_t start_idx = db->qstart[i];  //starting index of the query
        int64_t end_idx = db->qend[i];      //ending index of the query

        int32_t qlen = end_idx - start_idx; //query chunk length

        int8_t rna = core->opt.flag & SIGFISH_RNA; // if data is RNA

        float *query = (float *)malloc(sizeof(float)*qlen);
        MALLOC_CHK(query);

        for(int j=0;j<qlen;j++){
            if (!(core->opt.flag & SIGFISH_INV) && rna){ //id rna we must reverse the events
                query[qlen-1-j] = db->et[i].event[j+start_idx].mean;
            }
            else{
                query[j] = db->et[i].event[j+start_idx].mean;
            }
        }

        for(int j=0;j<core->ref->num_ref;j++){

            int32_t rlen =core->ref->ref_lengths[j];
            float *cost = (float *)malloc(sizeof(float) * qlen * rlen);
            MALLOC_CHK(cost);

            subsequence(query, core->ref->forward[j], qlen , rlen, cost);
            for(int k=(qlen-1)*rlen; k< qlen*rlen; k+=qlen){
                float min_score = INFINITY;
                int32_t min_pos = -1;
                for(int m=0; m<qlen && k+m<qlen*rlen; m++){
                    if(cost[k+m] < min_score){
                        min_score = cost[k+m];
                        min_pos = m+k;
                    }
                }
                update_aln2(aln, min_score, j, min_pos-(qlen-1)*rlen, '+', cost, qlen, rlen);
            }


            if (!rna) { //if DNA we must consider the reverse strand as well
                subsequence(query, core->ref->reverse[j], qlen , rlen, cost);

                for(int k=(qlen-1)*rlen; k< qlen*rlen; k+=qlen){
                    float min_score = INFINITY;
                    int32_t min_pos = -1;
                    for(int m=0; m<qlen && k+m<qlen*rlen; m++){
                        if(cost[k+m] < min_score){
                            min_score = cost[k+m];
                            min_pos = m+k;
                        }
                    }
                    update_aln2(aln, min_score, j, min_pos-(qlen-1)*rlen, '-', cost, qlen, rlen);
                }

            }

            free(cost);

        }

        free(query);

        db->aln[i].score = aln[SECONDARY_CAP-1].score;
        db->aln[i].score2 = aln[SECONDARY_CAP-2].score;
        db->aln[i].pos_st = aln[SECONDARY_CAP-1].d == '+' ? aln[SECONDARY_CAP-1].pos_st : core->ref->ref_lengths[aln[SECONDARY_CAP-1].rid] - aln[SECONDARY_CAP-1].pos_end  ;
        db->aln[i].pos_end = aln[SECONDARY_CAP-1].d == '+' ? aln[SECONDARY_CAP-1].pos_end : core->ref->ref_lengths[aln[SECONDARY_CAP-1].rid] - aln[SECONDARY_CAP-1].pos_st  ;

        db->aln[i].pos_st += core->ref->ref_st_offset[aln[SECONDARY_CAP-1].rid];
        db->aln[i].pos_end += core->ref->ref_st_offset[aln[SECONDARY_CAP-1].rid];
        db->aln[i].rid = aln[SECONDARY_CAP-1].rid;
        db->aln[i].d = aln[SECONDARY_CAP-1].d;

        int mapq=(int)round(500*(db->aln[i].score2-db->aln[i].score)/db->aln[i].score);
        if(mapq>60){
            mapq=60;
        }
        db->aln[i].mapq = mapq;

        free(aln);
    }

}


void dtw_cuda_db(core_t *core, db_t *db){

    //copy from RAM to GPU

    //call GPU kernel
    int32_t i=0;
    for (i = 0; i < db->n_rec; i++) {
        dtw_single2(core,db,i);
    }
    //Copy results back

    return;
}